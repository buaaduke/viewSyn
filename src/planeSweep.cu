#include "hip/hip_runtime.h"
#include <string>
#include <iostream>


//texture<uchar4, hipTextureType3D, hipReadModeElementType> layeredTex;
texture<float, hipTextureType3D, hipReadModeElementType> layeredTex;
texture<uchar4, hipTextureType3D, hipReadModeElementType> colorTex;


surface<void, cudaSurfaceType3D> colorTex_Surface;
surface<void, cudaSurfaceType2D> depthmap_Surface;
surface<void, cudaSurfaceType2D> depthmapView_Surface;

#define CUDA_SAFE_CALL(err) _CUDA_SAFE_CALL( err,__FILE__, __LINE__)

void _CUDA_SAFE_CALL( hipError_t err, std::string file = __FILE__, int line = __LINE__)
{
	if (err != hipSuccess) {
		std::cout<< hipGetErrorString( err ) << " in file: " << file << " at line: " << line << std::endl;
        //printf( "%s in %s at line %i\n", hipGetErrorString( err ),
          //      file.c_str(), line );
        exit( EXIT_FAILURE );
    }
}


__global__ void cudaProcess(unsigned char *out_array, int imageWidth, int imageHeight, int numOfImages, unsigned int numOfCandidatePlanes)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(x < imageWidth && y < imageHeight)
	{
		float cost = 1000000.0f;		
		int planeIndex; float dataCost = 0; 
		for(unsigned int i = 0; i<numOfCandidatePlanes; i++)
		{		
			dataCost = tex3D(layeredTex, x + 0.5, y + 0.5, i + 0.5);
			if(dataCost < cost)
			{
				cost = dataCost;
				planeIndex = i;
			}		
		}
		uchar4 pixelColor = tex3D(colorTex, x + 0.5, y + 0.5, planeIndex + 0.5);
		out_array[(y * imageWidth  + x ) * 4 + 0] = pixelColor.x;
		out_array[(y * imageWidth  + x ) * 4 + 1] = pixelColor.y;
		out_array[(y * imageWidth  + x ) * 4 + 2] = pixelColor.z;
		out_array[(y * imageWidth  + x ) * 4 + 3] = pixelColor.w;
	}
}

__global__ void findDepthMap(int imageWidth, int imageHeight, unsigned int numOfCandidatePlanes,
	float near, float far, float step)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(x < imageWidth && y < imageHeight)
	{
		float cost = 1000000.0f;
		float cost2nd = 1000000.0f;
		int planeIndex; float dataCost = 0; 
		int planeIndex2nd;
		for(unsigned int i = 0; i<numOfCandidatePlanes; i++)
		{		
			dataCost = tex3D(layeredTex, x + 0.5, y + 0.5, i + 0.5);
			
			if(dataCost < cost)
			{
				cost2nd = cost;	// the smallest cost replace the second smallest cost
				planeIndex2nd = planeIndex;
				//-----------------------------------------
				cost = dataCost;
				planeIndex = i;
			}
			else if( dataCost < cost2nd)
			{
				cost2nd = dataCost;
				planeIndex2nd = i;
			}
		}
		float depth;
		//if( (cost2nd - cost)/(cost2nd + 0.00001) < -0.99 && abs(planeIndex - planeIndex2nd)>1)	// the depth is not reliable
		//{			
			//printf("cost: %f\n", (cost2nd - cost)/(cost2nd + 0.00001));
			//planeIndex = numOfCandidatePlanes - 1;		// set the index to the last plane
			//depth = far;
		//	planeIndex = 1;
		//	depth = near + 0.02;
		//}
		//else
		//{
			float d = -1.0f + step * float( planeIndex + 1);
			depth = -2 * far * near/ (d * (far - near) - (far + near));
		//}
		//printf("%u \n", planeIndex);
		surf2Dwrite( planeIndex, depthmap_Surface, x * 4, y, hipBoundaryModeTrap);
		
		float normalizedDepth = 255.0f * (depth - near)/ (far - near);
		uchar4 depthValue = make_uchar4(normalizedDepth, normalizedDepth,normalizedDepth, 255);

		//surf2Dwrite( depthValue, depthmapView_Surface, x * 4, y, hipBoundaryModeTrap);
	}
}

__global__ void writeToSurfaceColor(int width, int height)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x; 
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if(x < width && y < height)
	{
		for(int i = 0; i<25; i++)
		{
			uchar4 d1;
			surf3Dread(&d1, colorTex_Surface, x*4, y, i);
			uchar4 data = make_uchar4(255.0f/ float(i)*2.0, 255, 0, 255);
			surf3Dwrite(data, colorTex_Surface, x * 4, y, i);
		}
	}

}


__global__ void writeDepthValue(int width, int height, float near, float far, float step)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	
	if(x < width && y < height)
	{
		int planeIdx;
		surf2Dread( &planeIdx, depthmap_Surface, x * 4, y,  hipBoundaryModeTrap);
		float d = -1.0f + step * float( planeIdx + 1);
		float depth = -2 * far * near/ (d * (far - near) - (far + near));
		float normalizedDepth = 255.0f * (depth - near)/ (far - near);
		uchar4 depthValue = make_uchar4(normalizedDepth, normalizedDepth,normalizedDepth, 255);

		surf2Dwrite( depthValue, depthmapView_Surface, x * 4, y, hipBoundaryModeTrap);
	}
}


void launchCudaGetDepthMap(hipArray *cost3D_CUDAArray, hipArray *depthmap_CUDAArray  , hipArray *depthmapView_CUDAArray,
	int imgWidth, int imgHeight, unsigned int numOfCandidatePlanes, float near, float far, float step)
{
	// bind texture and surface 
	CUDA_SAFE_CALL(hipBindTextureToArray(layeredTex, cost3D_CUDAArray));
 	layeredTex.normalized = false;
	CUDA_SAFE_CALL(cudaBindSurfaceToArray(depthmapView_Surface, depthmapView_CUDAArray));
	CUDA_SAFE_CALL(cudaBindSurfaceToArray(depthmap_Surface, depthmap_CUDAArray));

	

	// launch kernel
	int blockDimX = 16; int blockDimY = 16;
	dim3 block(blockDimX, blockDimY, 1); 	
    dim3 grid( (imgWidth+block.x - 1) / block.x, (imgHeight + block.y - 1) / block.y, 1);
	findDepthMap<<<grid, block >>>(imgWidth, imgHeight, numOfCandidatePlanes, near, far, step);

	CUDA_SAFE_CALL(hipUnbindTexture(layeredTex));

}

void launchCudaWriteDepthIndexToImage(hipArray *depthmap_CUDAArray, hipArray *depthmapView_CUDAArray, int width, int height, float near, float far, float step)
{
	CUDA_SAFE_CALL(cudaBindSurfaceToArray(depthmapView_Surface, depthmapView_CUDAArray));
	CUDA_SAFE_CALL(cudaBindSurfaceToArray(depthmap_Surface, depthmap_CUDAArray));
	int blockDimX = 16; int blockDimY = 16;
	dim3 block(blockDimX, blockDimY, 1); 	
    dim3 grid( (width+block.x - 1) / block.x, (height + block.y - 1) / block.y, 1);

	writeDepthValue<<<grid, block>>> (width, height, near, far , step);

}

void launchCudaProcess(hipArray *cost3D_CUDAArray, hipArray *color3D_CUDAArray, unsigned char *out_array, int imgWidth, int imgHeight, int numOfImages, unsigned int numOfCandidatePlanes)
{
	CUDA_SAFE_CALL(hipBindTextureToArray(layeredTex, cost3D_CUDAArray));
	layeredTex.normalized = false;

	CUDA_SAFE_CALL(hipBindTextureToArray(colorTex, color3D_CUDAArray));
	colorTex.normalized = false;

	//cudaBindSurfaceToArray(colorTex_Surface, color3D_CUDAArray);


	int blockDimX = 16; int blockDimY = 16;
	dim3 block(blockDimX, blockDimY, 1); 	
    dim3 grid( (imgWidth+block.x - 1) / block.x, (imgHeight + block.y - 1) / block.y, 1);

	cudaProcess<<<grid, block >>>(out_array, imgWidth, imgHeight, numOfImages, numOfCandidatePlanes);

	//writeToSurfaceColor<<<grid, block>>>( imgWidth, imgHeight);
	

	//CUDA_SAFE_CALL(hipUnbindTexture(layeredTex));

	if ( hipSuccess != hipGetLastError() )
	   printf( "Error!\n" );

	CUDA_SAFE_CALL(hipUnbindTexture(layeredTex));
	CUDA_SAFE_CALL(hipUnbindTexture(colorTex));

}


